
#include <hip/hip_runtime.h>
//#include "cuda.h"
//#define __launch_bounds__(...) __attribute__((launch_bounds(__VA_ARGS__)))

int hipConfigureCall(int gridSize, int blockSize);

//int cudaConfigureCall(dim3 gridSize, dim3 blockSize, size_t sharedSize = 0, cudaStream_t stream = 0);

__attribute__((global)) void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = 2;
	b = 7;
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
