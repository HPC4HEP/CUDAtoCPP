
#include <hip/hip_runtime.h>
//#include "cuda.h"
//#define __launch_bounds__(...) __attribute__((launch_bounds(__VA_ARGS__)))

int hipConfigureCall(int gridSize, int blockSize);

//int cudaConfigureCall(dim3 gridSize, dim3 blockSize, size_t sharedSize);

//int cudaConfigureCall(dim3 gridSize, dim3 blockSize, size_t sharedSize, cudaStream_t stream);

__attribute__((global)) void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = 2;
	b = 7;
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
