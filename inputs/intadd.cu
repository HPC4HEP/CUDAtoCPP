
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int*c){
	*c = *a + *b;
}

int main(void){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);
	
	hipStream_t st;

	hipMalloc(&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	

	a = 2;
	b = 7;
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	dim3 ggg(1,1,1);
	dim3 bbb(1,1,1);
	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
