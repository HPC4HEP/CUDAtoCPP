#include "hip/hip_runtime.h"
__host__ void vecAdd(double *a, double *b, double *c, int n)
{

	

  int id = blockIdx.x*blockDim.x+threadIdx.x;
  if(id<n){
    c[id] = a[id] + b[id];
  }



}

int main(){
{
  int n = 1000;
  double *h_a, *h_b, *h_c;
  double *d_a, *d_b, *d_c;
  size_t bytes = n*sizeof(double);
  h_a = (double*)malloc(bytes);
  h_b = (double*)malloc(bytes);
  h_c = (double*)malloc(bytes);
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  int i;
  for( i = 0; i < n; i++ ) {
    h_a[i] = sin(i)*sin(i);
    h_b[i] = cos(i)*cos(i);
  }
  hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
  int blockSize, gridSize;
  blockSize = 1024;
  gridSize = (int)ceil((float)n/blockSize);



  vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_a);
  free(h_b);
  free(h_c);
  return 0;
}
